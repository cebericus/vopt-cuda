#include "hip/hip_runtime.h"
/* 
   Original Author of gol_textual.c:
	Christopher Mitchell <chrism@lclark.edu>
   CUDAfied version by Michael Barger <mbarger@pdx.edu>
    for Homework 2 for CS510[GPU] (Prof Karavanic)
 */	

#include "gol_common2.h"


// The two boards -- host only needs one
unsigned char h_current[WIDTH * HEIGHT];
unsigned char *d_current;
unsigned char *d_next;

const dim3 gridDim(8, 8, 1);
const dim3 blocksDim(16, 16, 1); // 256 threads per block


extern "C" __global__ void step (const unsigned char *current, unsigned char *next) {
    // coordinates of the cell we're currently evaluating
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // offset index, neighbor coordinates, alive neighbor count
    int i, nx, ny, num_neighbors;
    
    const int offsets[8][2] = {{-1, 1},{0, 1},{1, 1},
                           {-1, 0},       {1, 0},
                           {-1,-1},{0,-1},{1,-1}};

    // count this cell's alive neighbors
    num_neighbors = 0;
    for (i=0; i<8; i++) {
        // To make the board torroidal, we use modular arithmetic to
        // wrap neighbor coordinates around to the other side of the
        // board if they fall off.
        nx = (x + offsets[i][0] + WIDTH) % WIDTH;
        ny = (y + offsets[i][1] + HEIGHT) % HEIGHT;
        num_neighbors += current[ny * WIDTH + nx]==1;
    }

    // apply the Game of Life rules to this cell
    next[y * WIDTH + x] = ((current[y * WIDTH + x] && num_neighbors==2) || num_neighbors==3);
}


void loop_func() {
    step<<<gridDim, blocksDim>>>(d_current, d_next);
    cudaCheckError("kernel execution");

    hipMemcpy(h_current, d_next, field_size, hipMemcpyDeviceToHost);
    cudaCheckError("Device->Host memcpy");

    hipMemcpy(d_current, d_next, field_size, hipMemcpyDeviceToDevice);
    cudaCheckError("Device->Device memcpy");
}


int main(void) {
	// allocate the device-side field arrays
	hipMalloc((void **)&d_current, field_size);
	hipMalloc((void **)&d_next, field_size);
	cudaCheckError("device memory allocation");

    // Initialize the host-side "current".
    fill_board(h_current);
    
    // copy host memory to device
    hipMemcpy(d_current, h_current, field_size, hipMemcpyHostToDevice);
    cudaCheckError("init array host->device copy");
    
    // run the simulation!
    animate(loop_func, h_current);

	// free device memory
	hipFree(d_current);
	hipFree(d_next);
    return 0;
}