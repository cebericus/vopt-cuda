/* Modified for CUDA - Cole Nelson - 2012 Apr 11 - mail@colenelson.net
 *
 * To compile for CUDA with debugging and gcc-4.4:
 * nvcc --compiler-bindir /opt/gcc44 -lX11 -g -G -o life_0 ./life_0.cu
 * 
 * Author: Christopher Mitchell <chrism@lclark.edu>
 * Date: 2011-08-10
 *
 * Compile with `gcc -lX11 gol.c`.
 */

#include <stdlib.h> 	// for rand
#include <string.h> 	// for memcpy
#include <stdio.h> 	// for printf
#include <X11/Xlib.h> 	// for the graphics

#include <hip/hip_runtime.h>		// "for speed"

#define WIDTH 800
#define HEIGHT 600


// The game board 
int curr_board[WIDTH * HEIGHT];

// set random starting points on game boarde
void fill_board( int *board ) {
    int i;
    for ( i = 0; i < WIDTH * HEIGHT; ++i )
        board[i] = rand() % 2;
}


// this kernel processes the neighborhood around an individual cell 
__global__ void step_kernel( int * cuda_curr_board, int * cuda_next_board ){
	
	// array initializer not allowed for __shared__
	__shared__ int offsets[8][2];
	
	//{-1, 1},{0, 1},{1, 1},
	offsets[0][0] = -1; offsets[0][1] = 1;
	offsets[1][0] = 0; offsets[1][1] = 1;
	offsets[2][0] = 1; offsets[2][1] = 1;
									
	//{-1, 0},       {1, 0},
	offsets[3][0] = -1; offsets[3][1] = 0;
	offsets[4][0] = 1; offsets[4][1] = 0;
	
	//{-1,-1},{0,-1},{1,-1}};
	offsets[5][0] = -1; offsets[5][1] = -1;
	offsets[6][0] = 0; offsets[6][1] = -1;
	offsets[7][0] = 1; offsets[7][1] = -1;

	// offset index, neighbor coordinates, living neighbor count
	int i, nx, ny, num_neighbors;

	// count this cell's living neighbors
	num_neighbors = 0;
	
	// calculate thread indices
	int x = ( blockIdx.x * blockDim.x ) + threadIdx.x;
	int y = ( blockIdx.y * blockDim.y ) + threadIdx.y;
	
	for( i = 0; i < 8; ++i ) {
		
		// To make the board torroidal, we use modular arithmetic to
		// wrap neighbor coordinates around to the other side of the
		// board if they fall off.
		nx = ( x + offsets[i][0] + WIDTH ) % WIDTH;
		ny = ( y + offsets[i][1] + HEIGHT ) % HEIGHT;
		
		if( cuda_curr_board[ny * WIDTH + nx] ) {
			++num_neighbors;
		}
	}

	// apply the Game of Life rules to this cell
	cuda_next_board[y * WIDTH + x] = 0;
	
	// TODO does this cause thread wave problems?
	if( ( cuda_curr_board[y * WIDTH + x] && ( num_neighbors == 2 ) ||
			num_neighbors == 3 ) ) {
		
		cuda_next_board[y * WIDTH + x] = 1;
		
		// supposedly, lot's of syncthreads are ok
		__syncthreads();
	}
	
	__syncthreads();
}


// creates an X11 window, sets up CUDA memory and enters an endless while(1)
// loop to run the game
void animate() {
    Display* display;
    display = XOpenDisplay(NULL);
    
    if( display == NULL ) {
        fprintf( stderr, "Could not open an X display.\n" );
        exit( -1 );
    }
    
    int screen_num = DefaultScreen( display );

    int black = BlackPixel( display, screen_num );
    int white = WhitePixel( display, screen_num );

    Window win = XCreateSimpleWindow( display,
            RootWindow( display, screen_num ),
            0, 0,
            WIDTH, HEIGHT,
            0,
            black, white );
    
    XStoreName( display, win, "The CUDA Game of Life" );

    XSelectInput( display, win, StructureNotifyMask );
    XMapWindow( display, win );
    while( 1 ) {
        XEvent e;
        XNextEvent( display, & e );
        if( e.type == MapNotify )
            break;	//<--------------------------internal break
    }

    GC gc = XCreateGC( display, win, 0, NULL );

    int x, y, n;						// display coords and points counter
    XPoint points[WIDTH * HEIGHT];		// display array
    
    /* begin setup of cuda specific host code */
    // simple error checking
    hipError_t err;
    
    // calculate the size of game board
    size_t size_board = WIDTH * HEIGHT * sizeof( int );
    
    // pointers to game boards on the device
    int *cuda_curr_board = NULL;
    int *cuda_next_board = NULL;
    
    // allocate memory on device: cudaError_t cudaMalloc( void ** devPtr, size_t size );
    err = hipMalloc( (void **) &cuda_curr_board, size_board );
    if( err != hipSuccess )
    	printf( "cudamalloc error: curr_board\n" );
    
    // allocate memory on device
    err = hipMalloc( (void **) &cuda_next_board, size_board );
    if( err != hipSuccess )
    	printf( "cudamalloc error: next_board\n" );
    
    // cuda dimensions
    dim3 threadsPerBlock( 16, 16 );
    dim3 numBlocks( WIDTH / threadsPerBlock.x, HEIGHT / threadsPerBlock.y );
    
    /* endless game loop */
    while( 1 ){
    	
        XClearWindow( display, win );
        
        // init counter for points array
        n = 0;
        
        // set the Xpoints for display from the game board array
        for( y = 0; y < HEIGHT; ++y ) {
            for( x = 0; x < WIDTH; ++x ) {
                if( curr_board[y * WIDTH + x] ) {
                    points[n].x = x;
                    points[n].y = y;
                    ++n;
                }
            }
        }
        
        // display current board state
        XDrawPoints( display, win, gc, points, n, CoordModeOrigin );
        XFlush( display );
        
        // copy game board to device: 
        // cudaError_t cudaMemcpy( void * dst, const void * src, size_t count, enum cudaMemcpyKind kind);
        err = hipMemcpy( cuda_curr_board, curr_board, size_board, hipMemcpyHostToDevice );
        if( err != hipSuccess )
        	printf( "hipMemcpy error: to device\n" );
        
        // calculate next game board
        step_kernel<<< numBlocks, threadsPerBlock >>>( cuda_curr_board, cuda_next_board );
        
        // copy game board from device to host board
        err = hipMemcpy( curr_board, cuda_next_board, size_board, hipMemcpyDeviceToHost );
        if( err != hipSuccess )
        	printf( "hipMemcpy error: from device\n" );
    }
    /* end endless game loop */
    
    // Note: unreachable - TODO need to test for a static board instead of while(1)
    hipFree( cuda_curr_board );
    hipFree( cuda_next_board );
    /* end of cuda specific host code */
}

// initializes the host side game board and calls animate() to initalize CUDA 
// device game boards and the X11 display
int main( void ) {
	
    // Initialize the global "current_board".
    fill_board( curr_board );
    
    // run game
    animate();

    return 0;
}


